#include "hip/hip_runtime.h"
#include "gpu_naive.h"


__global__
void matmul_naive_kernel(float* A, float* B, float* C, int M, int N,int K){
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if(row<M and col<N){
        float sum = 0.0f;
        for(int k=0;k<K;k++){
            sum+=A[row*K+k] * B[k*N+col];
        }
        C[row*N+col] = sum;
    }
}
void gpu_matmul_naive(const float* A, const float* B, float* C, int M, int N, int K){
    // allocate memory on the GPU
    float *d_A, *d_B, *d_C;
    size_t size_A = M * K * sizeof(float);
    size_t size_B = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);

    hipMalloc((void**)&d_A, size_A);
    hipMalloc((void**)&d_B, size_B);
    hipMalloc((void**)&d_C, size_C);

    hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice);

    dim3 blockDim(16,16);
    dim3 gridDim((N+15)/16, (M+15)/16);

    matmul_naive_kernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, N, K);
    hipDeviceSynchronize();

    hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}